#include "hip/hip_runtime.h"
#include "bcl_parser_cuda.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>

// Helper macro for CUDA error checking
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// CUDA kernel to decode BCL data for all clusters in parallel
__global__ void decode_bcl_kernel(
    const char** d_bcl_data,      // Device pointers to raw BCL data for each cycle
    const int* d_read_structure,  // Defines which cycle belongs to which read segment
    char* d_output_sequences,     // Output buffer for all decoded sequences concatenated
    char* d_output_qualities,     // Output buffer for all decoded quality scores concatenated
    int num_cycles,               // Total number of cycles
    int total_sequence_length,    // Total length of all read segments combined
    int num_clusters              // Total number of clusters (reads)
) {
    int cluster_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (cluster_idx >= num_clusters) {
        return;
    }

    static const char bases[] = {'A', 'C', 'G', 'T', 'N'};

    int output_position = 0;  // Track position in output sequence
    for (int cycle = 0; cycle < num_cycles; ++cycle) {
        // Each BCL file buffer contains data for this batch of clusters for one cycle
        // cluster_idx is local to this batch (0 to num_clusters-1)
        char bcl_byte = d_bcl_data[cycle][cluster_idx];

        // Decode base and quality
        // BCL encoding: bits 0-1: base (0=A, 1=C, 2=G, 3=T), bits 2-7: quality
        int base_idx = bcl_byte & 0x03;
        int quality_val = (bcl_byte >> 2) & 0x3F;

        char base = (quality_val == 0) ? 'N' : bases[base_idx];
        char quality_char = static_cast<char>(quality_val + 33); // Phred+33

        // Determine where to write the decoded base and quality
        int read_segment = d_read_structure[cycle];
        if (read_segment >= 0 && output_position < total_sequence_length) { // Negative values can be used to skip cycles if needed
            int output_idx = cluster_idx * total_sequence_length + output_position;
            d_output_sequences[output_idx] = base;
            d_output_qualities[output_idx] = quality_char;
            output_position++;
        }
    }
}

// Host function to manage BCL decoding on the GPU
void decode_bcl_data_cuda(
    const std::vector<char*>& h_bcl_data,
    const std::vector<size_t>& h_bcl_sizes,
    const std::vector<int>& h_read_structure,
    std::vector<Read>& reads,
    size_t num_clusters
) {
    if (h_bcl_data.empty() || num_clusters == 0) {
        return;
    }

    int num_cycles = h_bcl_data.size();
    
    // Calculate actual lengths of each segment first
    int read1_len = 0, index1_len = 0, index2_len = 0, read2_len = 0;
    for (int seg : h_read_structure) {
        if (seg == 0) read1_len++;
        else if (seg == 1) index1_len++;
        else if (seg == 2) index2_len++;
        else if (seg == 3) read2_len++;
    }
    
    // Total sequence length is the sum of all segments
    int total_sequence_length = read1_len + index1_len + index2_len + read2_len;
    
    std::cout << "Debug: num_cycles=" << num_cycles 
              << ", total_sequence_length=" << total_sequence_length 
              << " (R1=" << read1_len << ", I1=" << index1_len 
              << ", I2=" << index2_len << ", R2=" << read2_len << ")" << std::endl;
    std::cout << "Debug: h_read_structure size=" << h_read_structure.size() << std::endl;

    // --- Streaming approach: process clusters in GPU-sized batches ---
    // Calculate a safe batch size based on available GPU memory
    size_t free_mem = 0, total_mem = 0;
    // Prefer hipMemGetInfo; if unavailable, fall back to device properties (totalGlobalMem)
    hipError_t meminfo_err = hipMemGetInfo(&free_mem, &total_mem);
    if (meminfo_err != hipSuccess) {
        int dev = 0;
        hipDeviceProp_t prop{};
        hipError_t dev_err = hipGetDevice(&dev);
        hipError_t prop_err = (dev_err == hipSuccess) ? hipGetDeviceProperties(&prop, dev) : hipErrorUnknown;
        if (dev_err == hipSuccess && prop_err == hipSuccess && prop.totalGlobalMem > 0) {
            // Allow override of usable fraction via env var
            double frac = 0.60; // default to 60% of total mem for working set
            if (const char* env = std::getenv("CUDA_DEMUX_MEM_FRACTION")) {
                try {
                    double v = std::stod(env);
                    if (v > 0.05 && v <= 0.95) frac = v;
                } catch (...) {}
            }
            total_mem = static_cast<size_t>(prop.totalGlobalMem);
            free_mem  = static_cast<size_t>(prop.totalGlobalMem * frac);
            std::cerr << "Info: hipMemGetInfo unavailable (" << hipGetErrorString(meminfo_err)
                      << "); using totalGlobalMem with fraction " << frac << "." << std::endl;
        } else {
            // Final fallback: modest static budget
            total_mem = (size_t)3ULL * 1024ULL * 1024ULL * 1024ULL;
            free_mem  = (size_t)2ULL * 1024ULL * 1024ULL * 1024ULL;
            std::cerr << "Warning: Could not query device properties; using static memory estimates." << std::endl;
        }
    }

    // Bytes per cluster on device: one byte per cycle + two output bytes per cycle across segments
    // Approximate as: num_cycles (input) + 2*total_sequence_length (outputs)
    size_t bytes_per_cluster = static_cast<size_t>(num_cycles) + static_cast<size_t>(2 * total_sequence_length);
    // Additional overhead: pointer arrays and read structure
    // Overhead includes pointer arrays and a small fixed margin (~1MB)
    size_t overhead = static_cast<size_t>(num_cycles) * (sizeof(char*) + sizeof(int)) + (1ULL << 20);
    // Use 70% of free memory to be conservative
    size_t usable = static_cast<size_t>(free_mem * 0.70);
    size_t max_clusters_by_mem = usable > overhead && bytes_per_cluster > 0
        ? (usable - overhead) / bytes_per_cluster
        : 0;

    // Determine batch size. Prefer memory-derived estimate with sane bounds.
    size_t batch_size = 0;
    if (max_clusters_by_mem > 0) {
        batch_size = std::min<size_t>(num_clusters, max_clusters_by_mem);
    }
    // Lower bound to ensure progress even with tight memory
    const size_t kMinBatch = 16384ULL; // 16k clusters
    if (batch_size == 0) batch_size = std::min<size_t>(num_clusters, kMinBatch);
    // Optional override via env var
    if (const char* env = std::getenv("CUDA_DEMUX_BATCH_SIZE")) {
        try {
            size_t override = std::stoull(env);
            if (override > 0) batch_size = std::min<size_t>(num_clusters, override);
        } catch (...) {}
    }
    std::cout << "Decoding on GPU in batches of up to " << batch_size
              << " clusters (" << num_clusters << " total)." << std::endl;

    // Ensure inputs make sense
    for (int i = 0; i < num_cycles; ++i) {
        if (h_bcl_sizes[i] < num_clusters) {
            std::cerr << "Warning: cycle " << i << " buffer smaller than num_clusters (" << h_bcl_sizes[i] << " < " << num_clusters << ")" << std::endl;
        }
    }

    // Allocate device-side constant data used across batches
    char** d_bcl_data_ptrs = nullptr; // device array of per-cycle pointers
    int* d_read_structure = nullptr;
    CUDA_CHECK(hipMalloc(&d_bcl_data_ptrs, num_cycles * sizeof(char*)));
    CUDA_CHECK(hipMalloc(&d_read_structure, num_cycles * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_read_structure, h_read_structure.data(), num_cycles * sizeof(int), hipMemcpyHostToDevice));

    // Host output is assembled per batch directly into reads
    reads.resize(num_clusters);

    std::cout << "Decoding on GPU in batches of up to " << batch_size << " clusters (" << num_clusters << " total)." << std::endl;

    // Launch configuration
    int threads_per_block = 256;

    // Buffers allocated per batch
    std::vector<char*> d_bcl_data_buffers(num_cycles, nullptr);

    for (size_t start = 0; start < num_clusters; start += batch_size) {
        size_t this_batch = std::min(batch_size, num_clusters - start);
        
        std::cout << "Processing batch: start=" << start << ", this_batch=" << this_batch << std::endl;

        // Allocate per-cycle input buffers for this batch and copy slices
        for (int i = 0; i < num_cycles; ++i) {
            // Verify we have enough data in the source buffer
            if (h_bcl_sizes[i] < start + this_batch) {
                std::cerr << "ERROR: Cycle " << i << " buffer too small: " 
                          << h_bcl_sizes[i] << " < " << (start + this_batch) << std::endl;
                return;
            }
            CUDA_CHECK(hipMalloc(&d_bcl_data_buffers[i], this_batch * sizeof(char)));
            const char* h_src = h_bcl_data[i] + start;
            CUDA_CHECK(hipMemcpy(d_bcl_data_buffers[i], h_src, this_batch * sizeof(char), hipMemcpyHostToDevice));
        }
        std::cout << "Allocated " << num_cycles << " device buffers, each with " << this_batch << " bytes" << std::endl;

        // Update device pointer array
        CUDA_CHECK(hipMemcpy(d_bcl_data_ptrs, d_bcl_data_buffers.data(), num_cycles * sizeof(char*), hipMemcpyHostToDevice));

        // Allocate outputs for this batch
        char* d_output_sequences = nullptr;
        char* d_output_qualities = nullptr;
        size_t out_bytes = this_batch * static_cast<size_t>(total_sequence_length) * sizeof(char);
        CUDA_CHECK(hipMalloc(&d_output_sequences, out_bytes));
        CUDA_CHECK(hipMalloc(&d_output_qualities, out_bytes));

        // Launch kernel
        int blocks_per_grid = static_cast<int>((this_batch + threads_per_block - 1) / threads_per_block);
        std::cout << "Launching kernel: blocks=" << blocks_per_grid 
                  << ", threads_per_block=" << threads_per_block 
                  << ", num_clusters=" << static_cast<int>(this_batch) << std::endl;
        decode_bcl_kernel<<<blocks_per_grid, threads_per_block>>>(
            (const char**)d_bcl_data_ptrs,
            d_read_structure,
            d_output_sequences,
            d_output_qualities,
            num_cycles,
            total_sequence_length,
            static_cast<int>(this_batch)
        );
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Copy results back
        std::vector<char> h_output_sequences_batch(out_bytes);
        std::vector<char> h_output_qualities_batch(out_bytes);
        CUDA_CHECK(hipMemcpy(h_output_sequences_batch.data(), d_output_sequences, out_bytes, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_output_qualities_batch.data(), d_output_qualities, out_bytes, hipMemcpyDeviceToHost));

        // Free batch outputs on device
        hipFree(d_output_sequences);
        hipFree(d_output_qualities);

        // Assemble reads for this batch
        for (size_t j = 0; j < this_batch; ++j) {
            size_t read_idx = start + j;
            const char* seq_ptr = h_output_sequences_batch.data() + j * total_sequence_length;
            const char* qual_ptr = h_output_qualities_batch.data() + j * total_sequence_length;

            int offset = 0;
            reads[read_idx].sequence.assign(seq_ptr + offset, read1_len);
            reads[read_idx].quality.assign(qual_ptr + offset, read1_len);
            offset += read1_len;

            reads[read_idx].index1.assign(seq_ptr + offset, index1_len);
            offset += index1_len;

            reads[read_idx].index2.assign(seq_ptr + offset, index2_len);
            offset += index2_len;

            reads[read_idx].read2_sequence.assign(seq_ptr + offset, read2_len);
            reads[read_idx].read2_quality.assign(qual_ptr + offset, read2_len);
        }

        // Free per-cycle input buffers for this batch
        for (int i = 0; i < num_cycles; ++i) {
            hipFree(d_bcl_data_buffers[i]);
            d_bcl_data_buffers[i] = nullptr;
        }

        std::cout << "  Completed batch " << (start / batch_size + 1) << "/" << ((num_clusters + batch_size - 1) / batch_size)
                  << " (" << this_batch << " clusters)." << std::endl;
    }

    // Free persistent device resources
    hipFree(d_bcl_data_ptrs);
    hipFree(d_read_structure);

    std::cout << "Assembled " << reads.size() << " reads from GPU results (streamed)." << std::endl;
}
